#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <utility>
#include <cassert>
#include <cstdio>
#include "H5Cpp.h"

/* Geometry & parameters ******************************************************/

// Block size
constexpr size_t B0 = 32; // ideally = warp size, for coalesced read & write
constexpr size_t B1 = 4;
constexpr size_t B2 = 4;
constexpr size_t B3 = 2;
constexpr size_t blockSize = B0*B1*B2*B3;
// Maximal number of threads 32*4*4*2 = 1024
// Shared memory usage: 28kio including RNG state.

// Grid size
constexpr size_t G0 = 2;
constexpr size_t G1 = 8;
constexpr size_t G2 = 8;
constexpr size_t G3 = 16;
constexpr size_t gridSize = G0*G1*G2*G3;
  
// Lattice size
constexpr size_t N0 = B0*G0;
constexpr size_t N1 = B1*G1;
constexpr size_t N2 = B2*G2;
constexpr size_t N3 = B3*G3;

// Data array size (including ghost cells)
constexpr size_t M0 = N0+2;
constexpr size_t M1 = N1+2;
constexpr size_t M2 = N2+2;
constexpr size_t M3 = N3+2;
constexpr size_t M_count = M0*M1*M2*M3;
constexpr size_t M_bytes = M_count*sizeof(float);

// Lattice spacing
constexpr float a = 1.0f;

// Physical parameters
constexpr float m2 = 1.0f;
constexpr float lambda = 1.0f;

// Monte-Carlo parameters
constexpr unsigned int N_cor = 20;
constexpr unsigned int N_cf  = 100;
constexpr unsigned int N_th  = 10*N_cor;
constexpr float epsilon = 0.5f;

// Output
const H5std_string file_name("correlations.h5");
const H5std_string dataset_name("corr");

/******************************************************************************/

// Change in the action when φ(i) → φ(i) + ζ
// Idx: array index, including ghost cells
__device__ float delta_S_kin(float * f, const size_t Idx, const float zeta) {

  return a*a*zeta*( 4.0f*zeta + 8.0f*f[Idx]
                    - f[Idx+1]        - f[Idx-1]        // ± (1,0,0,0)
                    - f[Idx+M0]       - f[Idx-M0]       // ± (0,1,0,0)
                    - f[Idx+M0*M1]    - f[Idx-M0*M1]    // ± (0,0,1,0)
                    - f[Idx+M0*M1*M2] - f[Idx-M0*M1*M2] // ± (0,0,0,1)
                    );
}

__device__ float delta_S_free(float * f, const size_t Idx, const float zeta) {

  const float fi = f[Idx];
  const float delta_V = 0.5f*m2*zeta*(2.0f*fi+zeta);
  return delta_S_kin(f, Idx, zeta) + a*a*a*a*delta_V;
}

__device__ float delta_S_phi4(float * f, const size_t Idx, const float zeta) {

  const float fi = f[Idx];     // φi
  const float fiP = fi + zeta; // φi + ζ
  const float delta_V = 0.5f*m2*( fiP*fiP - fi*fi ) + 0.25f*lambda*( fiP*fiP*fiP*fiP - fi*fi*fi*fi );
  return delta_S_kin(f, Idx, zeta) + a*a*a*a*delta_V;
}


// Compute array index (includes ghost cells)
__device__ size_t array_idx(size_t Idx) {

  const size_t l = Idx / (N0*N1*N2);
  Idx -= l * N0*N1*N2;
  const size_t k = Idx / (N0*N1);
  Idx -= k * N0*N1;
  const size_t j = Idx / N0;
  Idx -= j * N0;

  return Idx+1 + M0*(j+1) + M0*M1*(k+1) + M0*M1*M2*(l+1);
}

template <float (*delta_S)(float*, const size_t, const float)>
__global__ void mc_kernel(float * lat, float * lo, hiprandState * states) {

  // Global thread index = lattice site
  const size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
  // Array index
  const size_t Idx = array_idx(tid);

  hiprandState state = states[tid];
  float zeta = (2.0f*hiprand_uniform(&state) - 1.0f) * epsilon; // ζ ∈ [-ε,+ε]

  // Compute change in the action due to variation ζ at size Idx
  const float delta_S_i = delta_S(lo, Idx, zeta);
  
  // Update the lattice depending on the variation ΔSi
  const float update = (float) (delta_S_i < 0.0f || (exp(-delta_S_i) > hiprand_uniform(&state)));
  // Is the above really branchless ?
  lat[Idx] += update * zeta;

  states[tid] = state;
}

// Initialize RNG state
__global__ void rng_init(hiprandState * states) {

  const size_t Idx = blockIdx.x * blockDim.x + threadIdx.x;
  hiprand_init((unsigned long long)clock() + Idx, 0, 0, &states[Idx]);
}

// Exchange 3D "faces" of the 4D lattice
// Face 0 (stride = 1)
__global__ void exchange_faces_0(float * lat) {

  const size_t I1 = blockIdx.x * blockDim.x + threadIdx.x + 1;
  const size_t I2 = blockIdx.y * blockDim.y + threadIdx.y + 1;
  const size_t I3 = blockIdx.z * blockDim.z + threadIdx.z + 1;
  const size_t Idx = M0*I1 + M0*M1*I2 + M0*M1*M2*I3;

  lat[Idx         ] = lat[Idx + N0];
  lat[Idx + (N0+1)] = lat[Idx +  1];
}

// Face 1 (stride = M0)
__global__ void exchange_faces_1(float * lat) {

  const size_t I0 = blockIdx.x * blockDim.x + threadIdx.x + 1;
  const size_t I2 = blockIdx.y * blockDim.y + threadIdx.y + 1;
  const size_t I3 = blockIdx.z * blockDim.z + threadIdx.z + 1;
  const size_t Idx = I0 + M0*M1*I2 + M0*M1*M2*I3;

  lat[Idx            ] = lat[Idx + M0*N1];
  lat[Idx + M0*(N1+1)] = lat[Idx + M0   ];
}

// Face 2 (stride = M0·M1)
__global__ void exchange_faces_2(float * lat) {

  const size_t I0 = blockIdx.x * blockDim.x + threadIdx.x + 1;
  const size_t I1 = blockIdx.y * blockDim.y + threadIdx.y + 1;
  const size_t I3 = blockIdx.z * blockDim.z + threadIdx.z + 1;
  const size_t Idx = I0 + M0*I1 + M0*M1*M2*I3;

  lat[Idx               ] = lat[Idx + M0*M1*N2];
  lat[Idx + M0*M1*(N2+1)] = lat[Idx + M0*M1   ];
}

// Face 3 (stride = M0·M1·M2)
__global__ void exchange_faces_3(float * lat) {

  const size_t I0 = blockIdx.x * blockDim.x + threadIdx.x + 1;
  const size_t I1 = blockIdx.y * blockDim.y + threadIdx.y + 1;
  const size_t I2 = blockIdx.z * blockDim.z + threadIdx.z + 1;
  const size_t Idx = I0 + M0*I1 + M0*M1*I2;

  lat[Idx                  ] = lat[Idx + M0*M1*M2*N3];
  lat[Idx + M0*M1*M2*(N3+1)] = lat[Idx + M0*M1*M2   ];
}

// Exchange all faces
__host__ void exchange_faces(float * lat) {

  exchange_faces_0<<<dim3(G1,G2,G3),dim3(B1,B2,B3)>>>(lat);
  exchange_faces_1<<<dim3(G0,G2,G3),dim3(B0,B2,B3)>>>(lat);
  exchange_faces_2<<<dim3(G0,G1,G3),dim3(B0,B1,B3)>>>(lat);
  exchange_faces_3<<<dim3(G0,G1,G2),dim3(B0,B1,B2)>>>(lat);
  hipDeviceSynchronize();
}

template <float (*delta_S)(float*, const size_t, const float)>
void mc_update(float* lat, float * lat_old, hiprandState * states) {

  mc_kernel<delta_S><<<gridSize,blockSize>>>(lat, lat_old, states);
  hipDeviceSynchronize();
  exchange_faces(lat);
  std::swap(lat, lat_old);
}

__global__ void time_slice_corr(float * lat, const size_t delta, float * corr) {

  const size_t I1 = blockIdx.x * blockDim.x + threadIdx.x;
  const size_t I2 = blockIdx.y * blockDim.y + threadIdx.y;
  const size_t I3 = blockIdx.z * blockDim.z + threadIdx.z;
  const size_t Idx = 1 + (I1+1)*M0 + (I2+1)*M0*M1 + (I3+1)*M0*M1*M2;

  float acc = 0.0f;
  for (size_t I0 = 0 ; I0 < N0 ; ++I0) {

    acc += lat[Idx+I0]*lat[Idx+(I0+delta)%N0];
  }

  corr[I1 + N1*I2 + N1*N2*I3] = acc / N0;
}

__host__ float reduce_3d(float * corr) {

  float acc = 0.0f;
  for (size_t I3 = 0 ; I3 < N3 ; ++I3)
    for (size_t I2 = 0 ; I2 < N2 ; ++I2)
      for (size_t I1 = 0 ; I1 < N1 ; ++I1) {

        acc += corr[I1 + N1*I2 + N1*N2*I3];
      }

  return acc / (N1*N2*N3);
}

__host__ void compute_correlations(float * lat, float * res, size_t run,
                                   float * corr_buf_h, float * corr_buf_d) {

  for (size_t delta = 0 ; delta < N0 ; ++delta) {

    time_slice_corr<<<dim3(B1,B2,B3),dim3(G1,G2,G3)>>>(lat, delta, corr_buf_d);
    hipDeviceSynchronize();
    hipMemcpy(corr_buf_h, corr_buf_d, N1*N2*N3*sizeof(float), hipMemcpyDeviceToHost);
    res[run + N_cf*delta] = reduce_3d(corr_buf_h);
  }
}

int write_correlations(float * corr) {

  try {

    H5::H5File file(file_name, H5F_ACC_TRUNC);
    hsize_t dims[2] = {N_cf, N0};
    H5::DataSpace dataspace(2, dims);
    auto dataset = file.createDataSet(dataset_name, H5::PredType::NATIVE_FLOAT, dataspace);
    dataset.write(corr, H5::PredType::NATIVE_FLOAT);
    dataset.close();
    file.close();
  }

  // catch failure caused by the H5File operations
  catch(H5::FileIException error)
    {
      error.printError();
      return -1;
    }
  // catch failure caused by the DataSet operations
  catch(H5::DataSetIException error)
    {
      error.printError();
      return -1;
    }
  // catch failure caused by the DataSpace operations
  catch(H5::DataSpaceIException error)
    {
      error.printError();
      return -1;
    }
  // catch failure caused by the DataSpace operations
  catch(H5::DataTypeIException error)
    {
      error.printError();
      return -1;
    }
  return 0;  // successfully terminated  
}

int write_configuration(float * lat) {

  H5::H5File file("configuration.h5", H5F_ACC_TRUNC);
  hsize_t dims[4]   = {N0,N1,N2,N3};
  hsize_t offset[4] = { 1, 1, 1, 1};
  H5::DataSpace dataspace(4, dims);
  // dataspace.selectHyperslab();   <----------------------- TO-DO
}

constexpr auto dS = delta_S_free;

__host__ void mc_average() {

  fprintf(stderr, "Lattice: (%d,%d,%d,%d)\n", N0, N1, N2, N3);
  fprintf(stderr, "Array:   (%d,%d,%d,%d)\n", M0, M1, M2, M3);
  fprintf(stderr, "M_count = %d\n", M_count);
  
  fprintf(stderr, "Allocating lattice arrays...\n");
  // Allocate lattice on device (double buffered)
  float * lat     = nullptr;
  float * lat_old = nullptr;
  fprintf(stderr, "Requesting 2×%d bytes...", M_bytes);
  hipMalloc(&lat    , M_bytes);
  hipMalloc(&lat_old, M_bytes);
  fprintf(stderr, " done.\n");
  fprintf(stderr, "Memset'ting to 0...");
  hipMemset(lat    , 0., M_count);
  hipMemset(lat_old, 0., M_count);
  fprintf(stderr, " done.\n");

  // Seed rng on each thread
  fprintf(stderr, "Allocating RNG...\n");
  fprintf(stderr, "Requesting %d bytes...", M_count*sizeof(hiprandState));
  hiprandState * states;
  hipMalloc(&states, M_count*sizeof(hiprandState));
  fprintf(stderr, " done.\n");
  fprintf(stderr, "Initializing RNG...");
  rng_init<<<gridSize,blockSize>>>(states);
  hipDeviceSynchronize();
  fprintf(stderr, " done.\n");

  // Allocate memory used to store correlation data
  // Host-side buffer
  float * corr_buf_h = (float*) calloc(N1*N2*N3, sizeof(float));
  assert(corr_buf_h);
  // Device-side buffer
  float * corr_buf_d = nullptr;
  hipMalloc(&corr_buf_d, N1*N2*N3*sizeof(float));
  // Array storing the final results
  float * corr = (float*) calloc(N0*N_cf, sizeof(float));
  assert(corr);

  // Thermalize lattice
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  fprintf(stderr, "Thermalizing lattice...");
  hipEventRecord(start);
  for (size_t i = 0 ; i < N_th ; ++i) {
    mc_update<dS>(lat, lat_old, states);
  }
  hipEventRecord(stop);
  hipEventSynchronize(stop);
  float ms;
  hipEventElapsedTime(&ms, start, stop);
  fprintf(stderr, " done in %fs.\n", 1e-3*ms);

  // Run Metropolis algorithm
  fprintf(stderr, "Running MC...");
  hipEventRecord(start);
  for (size_t i = 0 ; i < N_cf ; ++i) {
    // Drop N_cor iterations to damp correlations between successive configurations.
    for (size_t j = 0 ; j < N_cor ; ++j) {
      mc_update<dS>(lat, lat_old, states);
    }
    fprintf(stderr, " %d", i);
    // Compute the Euclidean time correlations within one configuration.
    // compute_correlations(lat_old, corr, i, corr_buf_h, corr_buf_d);
  }
  hipEventRecord(stop);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&ms, start, stop);
  fprintf(stderr, " done in %fs.\n", 1e-3*ms);

  // Write output to file
  fprintf(stderr, "Writing to file...");
  write_correlations(corr);
  fprintf(stderr, " done.\n");
  
  // Finalization
  // ============

  fprintf(stderr, "Finalization...");
  // Free device memory
  hipFree(lat);
  hipFree(lat_old);
  hipFree(states);
  hipFree(corr_buf_d);
  lat        = nullptr;
  lat_old    = nullptr;
  states     = nullptr;
  corr_buf_d = nullptr;

  // Free host memory
  free(corr_buf_h);
  free(corr);
  corr_buf_h = nullptr;
  corr       = nullptr;
  fprintf(stderr, " done.\n");
}

void generate_single_conf() {

  fprintf(stderr, "Lattice: (%d,%d,%d,%d)\n", N0, N1, N2, N3);
  fprintf(stderr, "Array:   (%d,%d,%d,%d)\n", M0, M1, M2, M3);
  fprintf(stderr, "M_count = %d\n", M_count);
  
  fprintf(stderr, "Allocating lattice arrays...\n");
  // Allocate lattice on device (double buffered)
  float * lat     = nullptr;
  float * lat_old = nullptr;
  fprintf(stderr, "Requesting 2×%d bytes...", M_bytes);
  hipMalloc(&lat    , M_bytes);
  hipMalloc(&lat_old, M_bytes);
  fprintf(stderr, " done.\n");
  fprintf(stderr, "Memset'ting to 0...");
  hipMemset(lat    , 0., M_count);
  hipMemset(lat_old, 0., M_count);
  fprintf(stderr, " done.\n");

  // Seed rng on each thread
  fprintf(stderr, "Allocating RNG...\n");
  fprintf(stderr, "Requesting %d bytes...", M_count*sizeof(hiprandState));
  hiprandState * states;
  hipMalloc(&states, M_count*sizeof(hiprandState));
  fprintf(stderr, " done.\n");
  fprintf(stderr, "Initializing RNG...");
  rng_init<<<gridSize,blockSize>>>(states);
  hipDeviceSynchronize();
  fprintf(stderr, " done.\n");

  // Thermalize lattice
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  fprintf(stderr, "Thermalizing lattice...");
  hipEventRecord(start);
  for (size_t i = 0 ; i < N_th ; ++i) {
    mc_update<dS>(lat, lat_old, states);
  }
  hipEventRecord(stop);
  hipEventSynchronize(stop);
  float ms;
  hipEventElapsedTime(&ms, start, stop);
  fprintf(stderr, " done in %fs.\n", 1e-3*ms);

  // Write result to file
  write_configuration(lat_old);

  fprintf(stderr, "Finalization...");
  // Free device memory
  hipFree(lat);
  hipFree(lat_old);
  hipFree(states);
  lat     = nullptr;
  lat_old = nullptr;
  states  = nullptr;
  fprintf(stderr, " done.\n");
}

__host__ int main() {

  //generate_single_conf();
  mc_average();

  return 0;
}
