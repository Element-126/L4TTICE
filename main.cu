#include "hip/hip_runtime.h"
// CUDA
#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
// Standard library
#include <utility>
#include <cassert>
#include <cstdio>
// HDF5
#include "H5Cpp.h"
// CUB
#include <hipcub/hipcub.hpp>

/******************************************************************************/

// Geometry & parameters
// =====================

// Block size
constexpr size_t B0 = 8;
constexpr size_t Bi = 8;
// Number of threads 8³ = 512
// Loop over the last dimension
// Shared memory usage: 44000o including halos.
// Then grid-stride loop to reuse the RNG state

// Grid size
constexpr size_t G0 = 1;
constexpr size_t Gi = 2;
  
// Lattice size
constexpr size_t N0 = B0*G0;
constexpr size_t Ni = Bi*Gi;

// Data array size (including ghost cells)
constexpr size_t M0 = N0+2;
constexpr size_t Mi = Ni+2;
constexpr size_t M_count = M0*Mi*Mi*Mi;
constexpr size_t M_bytes = M_count*sizeof(float);
// Strides
constexpr size_t S1 = M0;
constexpr size_t S2 = M0*Mi;
constexpr size_t S3 = M0*Mi*Mi;

// Physical parameters
constexpr float m2 = -1.0f;
constexpr float lambda = 0.01f;

// Monte-Carlo parameters
constexpr float epsilon = 1.0f;

// Output
const H5std_string file_name("correlations.h5");
const H5std_string dataset_name("corr");

/******************************************************************************/

// Variation of the action
// =======================

// Change in the action when φ(i) → φ(i) + ζ
// Idx: array index, including ghost cells
__device__ float delta_S_kin(float * f, const size_t Idx, const float zeta, const float a) {

  return a*a*zeta*( 4.0f*zeta + 8.0f*f[Idx]
                    - f[Idx+1 ] - f[Idx-1 ] // ± (1,0,0,0)
                    - f[Idx+S1] - f[Idx-S1] // ± (0,1,0,0)
                    - f[Idx+S2] - f[Idx-S2] // ± (0,0,1,0)
                    - f[Idx+S3] - f[Idx-S3] // ± (0,0,0,1)
                    );
}

// Free field: V(φ) = ½m²φ²
__device__ float delta_S_free(float * f, const size_t Idx, const float zeta, const float a) {

  const float fi = f[Idx];
  const float delta_V = 0.5f*m2*zeta*(2.0f*fi+zeta);
  return delta_S_kin(f, Idx, zeta, a) + a*a*a*a*delta_V;
}

// Interacting field: V(φ) = ½m²φ² + ¼λφ⁴
__device__ float delta_S_phi4(float * f, const size_t Idx, const float zeta, const float a) {

  const float fi = f[Idx];     // φi
  const float fiP = fi + zeta; // φi + ζ
  const float delta_V = 0.5f*m2*( fiP*fiP - fi*fi ) + 0.25f*lambda*( fiP*fiP*fiP*fiP - fi*fi*fi*fi );
  return delta_S_kin(f, Idx, zeta, a) + a*a*a*a*delta_V;
}

// Choice of the action used in the simulation
constexpr auto dS = delta_S_phi4;

/******************************************************************************/

// Kernels
// =======

// Main kernels, performing one Monte-Carlo iteration on either black or white indices.

/*  MC iteration over "black" indices
 *
 *  Blocksize should be (B0/2,Bi,Bi) and stride Bi
 *  Gridsize should be (G0,Gi,Gi) and grid stride Gi
 */ 
template<float (*delta_S)(float*, const size_t, const float, const float)>
__global__ void mc_update_black(float * lat, hiprandState * states, const float a) {

  // Global thread index
  const size_t t0 = blockIdx.x * blockDim.x + threadIdx.x;
  const size_t t1 = blockIdx.y * blockDim.y + threadIdx.y;
  const size_t t2 = blockIdx.z * blockDim.z + threadIdx.z;

  // Linear thread index
  const size_t tid = t0 + (N0>>1)*t1 + (N0*Ni>>1)*t2;
  
  auto state = states[tid];

  /*  Indices, assuming dimension 0 is even
   *
   *  Physical index: 2·t0 + N0·t1 + N0·N1·t2 + N0·N1·N2·t3 + parity of (t1+t2+t3)
   *  Ex: 4×4 lattice
   *      +–––––––––+
   *      | 0 · 4 · |
   *      | · 2 · 6 |
   *      | 1 · 5 · |
   *      | · 3 · 7 |
   *      +–––––––––+
   *
   *  Array index:    2·t0+1 + M0*(t1+1) + M0·M1·(t2+1) + M0·M1·M2·(t3+1) + parity of (t1+t2+t3)
   *  Ex: 4×4 lattice
   *
   *      |  halos  |
   *      v         v 
   *    +–––––––––––––+
   *    | × · × · × · | <– halo
   *    | · 0 · 4 · × |
   *    | × · 2 · 6 · |
   *    | · 1 · 5 · × |
   *    | × · 3 · 7 · |
   *    | · × · × · × | <– halo
   *    +–––––––––––––+
   */

  // Grid stride loop in direction 3
  for (size_t g3 = 0 ; g3 < Gi ; ++g3) {

    // Small loop in direction 3
    for (size_t b3 = 0 ; b3 < Bi ; ++b3) {

      const size_t t3 = g3*Bi+b3;
      
      // Array index (TODO: move this outside of the loop)
      const size_t parity = (t1 + t2 + t3) & 1; // 0 if t1+t2+t3 even, 1 otherwise
      const size_t Idx = 2*t0+1 + S1*(t1+1) + S2*(t2+1) + S3*(t3+1) + parity;

      const float zeta = (2.0f*hiprand_uniform(&state) - 1.0f) * epsilon; // ζ ∈ [-ε,+ε]
      // Compute change in the action due to variation ζ at site Idx
      const float delta_S_i = delta_S(lat, Idx, zeta, a);

      // Update the lattice depending on the variation ΔSi
      const float update = (float) (delta_S_i < 0.0f || (exp(-delta_S_i) > hiprand_uniform(&state)));
      // TODO - Is the above really branchless ?
      lat[Idx] += update * zeta;
    }
  }

  // Write RNG state back to global memory
  states[tid] = state;
}

/*  MC iteration over "white" indices
 *
 *  Same grid and block sizes as for black indices.
 */
template<float (*delta_S)(float*, const size_t, const float, const float)>
__global__ void mc_update_white(float * lat, hiprandState * states, const float a) {

  // Global thread index
  const size_t t0 = blockIdx.x * blockDim.x + threadIdx.x;
  const size_t t1 = blockIdx.y * blockDim.y + threadIdx.y;
  const size_t t2 = blockIdx.z * blockDim.z + threadIdx.z;

  // Linear thread index
  const size_t tid = t0 + (N0>>1)*t1 + (N0*Ni>>1)*t2;
  
  auto state = states[tid];

  // Grid stride loop in direction 3
  for (size_t g3 = 0 ; g3 < Gi ; ++g3) {

    // Small loop in direction 3
    for (size_t b3 = 0 ; b3 < Bi ; ++b3) {

      const size_t t3 = g3*Bi+b3;
      const size_t parity = (t1 + t2 + t3) & 1; // 0 if t1+t2+t3 even, 1 otherwise
      // Main difference with "black" indices: opposite parity
      const size_t Idx = 2*t0+1 + S1*(t1+1) + S2*(t2+1) + S3*(t3+1) + !parity;

      const float zeta = (2.0f*hiprand_uniform(&state) - 1.0f) * epsilon; // ζ ∈ [-ε,+ε]
      // Compute change in the action due to variation ζ at site Idx
      const float delta_S_i = delta_S(lat, Idx, zeta, a);

      // Update the lattice depending on the variation ΔSi
      const float update = (float) (delta_S_i < 0.0f || (exp(-delta_S_i) > hiprand_uniform(&state)));
      // Is the above really branchless ?
      lat[Idx] += update * zeta;
    }
  }

  // Write RNG state back to global memory
  states[tid] = state;
}

/*
 * Initialize RNG state
 *
 * Grid size: (G0,Gi,Gi)
 * Block size: (N0/2,Ni,Ni)
 */
__global__ void rng_init(unsigned long long seed, hiprandState * states) {

  const size_t I0 = blockIdx.x * blockDim.x + threadIdx.x;
  const size_t I1 = blockIdx.y * blockDim.y + threadIdx.y;
  const size_t I2 = blockIdx.z * blockDim.z + threadIdx.z;
  const size_t Idx = I0 + (N0>>1)*I1 + (N0*Ni>>1)*I2;
  hiprand_init(seed, Idx, 0, &states[Idx]);
}

/*
 * Set the 3d halos to zero so they do not affect the reduction
 *
 * Can be undone by calling exchange_faces
 */
__global__ void erase_halos_0(float * lat) {

  const size_t I1 = blockIdx.x * blockDim.x + threadIdx.x + 1;
  const size_t I2 = blockIdx.y * blockDim.y + threadIdx.y + 1;
  const size_t I3 = blockIdx.z * blockDim.z + threadIdx.z + 1;
  const size_t Idx = S1*I1 + S2*I2 + S3*I3;

  lat[Idx         ] = 0.0f;
  lat[Idx + (N0+1)] = 0.0f;
}

__global__ void erase_halos_1(float * lat) {

  const size_t I0 = blockIdx.x * blockDim.x + threadIdx.x + 1;
  const size_t I2 = blockIdx.y * blockDim.y + threadIdx.y + 1;
  const size_t I3 = blockIdx.z * blockDim.z + threadIdx.z + 1;
  const size_t Idx = I0 + S2*I2 + S3*I3;

  lat[Idx            ] = 0.0f;
  lat[Idx + S1*(Ni+1)] = 0.0f;
}

__global__ void erase_halos_2(float * lat) {

  const size_t I0 = blockIdx.x * blockDim.x + threadIdx.x + 1;
  const size_t I1 = blockIdx.y * blockDim.y + threadIdx.y + 1;
  const size_t I3 = blockIdx.z * blockDim.z + threadIdx.z + 1;
  const size_t Idx = I0 + S1*I1 + S3*I3;

  lat[Idx            ] = 0.0f;
  lat[Idx + S2*(Ni+1)] = 0.0f;
}

__global__ void erase_halos_3(float * lat) {

  const size_t I0 = blockIdx.x * blockDim.x + threadIdx.x + 1;
  const size_t I1 = blockIdx.y * blockDim.y + threadIdx.y + 1;
  const size_t I2 = blockIdx.z * blockDim.z + threadIdx.z + 1;
  const size_t Idx = I0 + S1*I1 + S2*I2;

  lat[Idx            ] = 0.0f;
  lat[Idx + S3*(Ni+1)] = 0.0f;
}

__host__ void erase_halos(float * lat) {

  erase_halos_0<<<dim3(Gi,Gi,Gi),dim3(Bi,Bi,Bi)>>>(lat);
  erase_halos_1<<<dim3(G0,Gi,Gi),dim3(B0,Bi,Bi)>>>(lat);
  erase_halos_2<<<dim3(G0,Gi,Gi),dim3(B0,Bi,Bi)>>>(lat);
  erase_halos_3<<<dim3(G0,Gi,Gi),dim3(B0,Bi,Bi)>>>(lat);
}

/******************************************************************************/

// Exchange of the 3d "faces" of the 4d lattice
// ============================================

// Face 0 (stride = 1)
__global__ void update_halos_0(float * lat) {

  const size_t I1 = blockIdx.x * blockDim.x + threadIdx.x + 1;
  const size_t I2 = blockIdx.y * blockDim.y + threadIdx.y + 1;
  const size_t I3 = blockIdx.z * blockDim.z + threadIdx.z + 1;
  const size_t Idx = S1*I1 + S2*I2 + S3*I3;

  lat[Idx         ] = lat[Idx + N0];
  lat[Idx + (N0+1)] = lat[Idx +  1];
}

// Face 1 (stride S1 = M0)
__global__ void update_halos_1(float * lat) {

  const size_t I0 = blockIdx.x * blockDim.x + threadIdx.x + 1;
  const size_t I2 = blockIdx.y * blockDim.y + threadIdx.y + 1;
  const size_t I3 = blockIdx.z * blockDim.z + threadIdx.z + 1;
  const size_t Idx = I0 + S2*I2 + S3*I3;

  lat[Idx            ] = lat[Idx + S1*Ni];
  lat[Idx + S1*(Ni+1)] = lat[Idx + S1   ];
}

// Face 2 (stride S2 = M0·M1)
__global__ void update_halos_2(float * lat) {

  const size_t I0 = blockIdx.x * blockDim.x + threadIdx.x + 1;
  const size_t I1 = blockIdx.y * blockDim.y + threadIdx.y + 1;
  const size_t I3 = blockIdx.z * blockDim.z + threadIdx.z + 1;
  const size_t Idx = I0 + S1*I1 + S3*I3;

  lat[Idx            ] = lat[Idx + S2*Ni];
  lat[Idx + S2*(Ni+1)] = lat[Idx + S2   ];
}

// Face 3 (stride S3 = M0·M1·M2)
__global__ void update_halos_3(float * lat) {

  const size_t I0 = blockIdx.x * blockDim.x + threadIdx.x + 1;
  const size_t I1 = blockIdx.y * blockDim.y + threadIdx.y + 1;
  const size_t I2 = blockIdx.z * blockDim.z + threadIdx.z + 1;
  const size_t Idx = I0 + S1*I1 + S2*I2;

  lat[Idx            ] = lat[Idx + S3*Ni];
  lat[Idx + S3*(Ni+1)] = lat[Idx + S3   ];
}

// Exchange all faces
__host__ void update_halos(float * lat) {

  update_halos_0<<<dim3(Gi,Gi,Gi),dim3(Bi,Bi,Bi)>>>(lat);
  update_halos_1<<<dim3(G0,Gi,Gi),dim3(B0,Bi,Bi)>>>(lat);
  update_halos_2<<<dim3(G0,Gi,Gi),dim3(B0,Bi,Bi)>>>(lat);
  update_halos_3<<<dim3(G0,Gi,Gi),dim3(B0,Bi,Bi)>>>(lat);
}

/******************************************************************************/

// Host-side logic
// ===============

// Perform one Monte-Carlo iteration
template <float (*delta_S)(float*, const size_t, const float, const float a)>
void mc_update(float* lat, hiprandState * states, const float a) {

  mc_update_black<delta_S><<<dim3(G0,Gi,Gi),dim3(B0/2,Bi,Bi)>>>(lat, states, a);
  mc_update_white<delta_S><<<dim3(G0,Gi,Gi),dim3(B0/2,Bi,Bi)>>>(lat, states, a);
  update_halos(lat);
}

// Resource management
// -------------------

__host__ float* new_lattice() {

  float * lat = nullptr;

  fprintf(stderr, "Lattice: (%d,%d,%d,%d)\n", N0, Ni, Ni, Ni);
  fprintf(stderr, "Array:   (%d,%d,%d,%d)\n", M0, Mi, Mi, Mi);
  fprintf(stderr, "M_count = %d\n", M_count);
  
  fprintf(stderr, "Allocating lattice array...\n");
  // Allocate lattice on device
  fprintf(stderr, "Requesting %d bytes...", M_bytes);
  assert(hipMalloc(&lat, M_bytes) == hipSuccess);
  fprintf(stderr, " done.\n");
  fprintf(stderr, "Memset'ting to 0...");
  assert(hipMemset(lat, 0.0f, M_count) == hipSuccess);
  fprintf(stderr, " done.\n");

  return lat;
}

__host__ void delete_lattice(float ** lat) {

  assert(hipFree(*lat) == hipSuccess);
  *lat = nullptr;
}

__host__ hiprandState* new_rng() {

  hiprandState * states;

  // Seed RNG on each thread
  fprintf(stderr, "Allocating RNG...\n");
  fprintf(stderr, "Requesting %d bytes...", N0/2*Ni*Ni*sizeof(hiprandState));
  assert(hipMalloc(&states, N0/2*Ni*Ni*sizeof(hiprandState)) == hipSuccess);
  fprintf(stderr, " done.\n");
  fprintf(stderr, "Initializing RNG...");
  rng_init<<<dim3(G0,Gi,Gi),dim3(B0/2,Bi,Bi)>>>(clock(), states);
  fprintf(stderr, " done.\n");

  return states;
}

__host__ void delete_rng(hiprandState ** states) {

  assert(hipFree(*states) == hipSuccess);
  *states = nullptr;
}

// Main algorithm
// --------------

// Compute the space-average of the time-slice correlator value over many configurations.
__host__ void mc_average(const size_t N_cf, const size_t N_cor, const size_t N_th, const float a) {

  auto lat = new_lattice();
  auto states = new_rng();

  // Allocate memory used to store correlation data
  // Host-side buffer
  float * corr_buf_h = (float*) calloc(Ni*Ni*Ni, sizeof(float));
  assert(corr_buf_h);
  // Device-side buffer
  float * corr_buf_d = nullptr;
  assert(hipMalloc(&corr_buf_d, Ni*Ni*Ni*sizeof(float)) == hipSuccess);
  // Array storing the final results
  float * corr = (float*) calloc(N0*N_cf, sizeof(float));
  assert(corr);

  // Thermalize lattice
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  fprintf(stderr, "Thermalizing lattice...");
  hipEventRecord(start);
  for (size_t i = 0 ; i < N_th ; ++i) {
    mc_update<dS>(lat, states, a);
  }
  hipEventRecord(stop);
  hipEventSynchronize(stop);
  float ms;
  hipEventElapsedTime(&ms, start, stop);
  fprintf(stderr, " done in %fs.\n", 1e-3*ms);

  // Run Metropolis algorithm
  fprintf(stderr, "Running MC...");
  hipEventRecord(start);
  for (size_t i = 0 ; i < N_cf ; ++i) {
    // Drop N_cor iterations to damp correlations between successive configurations.
    for (size_t j = 0 ; j < N_cor ; ++j) {
      mc_update<dS>(lat, states, a);
    }
    fprintf(stderr, " %d", i);
    // Compute the euclidean time correlations within one configuration.
  }
  hipEventRecord(stop);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&ms, start, stop);
  fprintf(stderr, " done in %fs.\n", 1e-3*ms);

  // Write output to file
  // fprintf(stderr, "Writing to file...");
  // write_correlations(corr);
  // fprintf(stderr, " done.\n");
  
  // Finalization
  // ============

  fprintf(stderr, "Finalization...");

  // Free device memory
  delete_lattice(&lat);
  delete_rng(&states);
  hipFree(corr_buf_d);
  corr_buf_d = nullptr;

  // Free host memory
  free(corr_buf_h);
  free(corr);
  corr_buf_h = nullptr;
  corr       = nullptr;
  fprintf(stderr, " done.\n");
}

void thermalize_conf(float * lat, hiprandState * states, const float a, const size_t N_th,
                     const bool verbose = false, const size_t poll = 1) {

  // Thermalize lattice
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  fprintf(stderr, "Thermalizing lattice...");
  hipEventRecord(start);
  for (size_t i = 1 ; i <= N_th ; ++i) {
    mc_update<dS>(lat, states, a);
    if (verbose && i % poll == 0) {
      fprintf(stderr, " %llu", i);
    }
  }
  hipEventRecord(stop);
  hipEventSynchronize(stop);
  float ms;
  hipEventElapsedTime(&ms, start, stop);
  fprintf(stderr, " done in %fs.\n", 1e-3*ms);
}

/*
 * Generate N_cf configurations, thermalize them and compute their means.
 *
 * Mandatory parameters:
 *   N_cf = number of configurations to generate
 *   N_th = number of MC iterations to thermalize the lattice
 *   a    = lattice spacing
 *
 * Optional parameters:
 *   verbose = whether to print the current status to stderr
 *   poll_th = number of MC updates between status updates
 */
void mc_mean(const size_t N_cf, const size_t N_th, const float a,
             const bool verbose = false, const size_t poll_th = 500) {

  // Allocate resources for the simulation
  auto lat = new_lattice();
  auto states = new_rng();

  //Prepare resources for CUB
  float * sum_d = nullptr;
  assert(hipMalloc(&sum_d, sizeof(float)) == hipSuccess);
  void * cub_tmp_storage = nullptr;
  size_t cub_tmp_bytes = 0;
  hipcub::CachingDeviceAllocator g_allocator(true);
  float * sum_h = (float*) malloc(sizeof(float));
  assert(sum_h);
  // Call once to initialize resources
  CubDebugExit(hipcub::DeviceReduce::Sum(cub_tmp_storage, cub_tmp_bytes, lat, sum_d, M_count));
  CubDebugExit(g_allocator.DeviceAllocate(&cub_tmp_storage, cub_tmp_bytes));

  for (size_t k = 1 ; k <= N_cf ; ++k) {

    // Thermalize the configuration
    thermalize_conf(lat, states, a, N_th, verbose, poll_th);
    // Erase the halos in order not to interfere with the summation
    erase_halos(lat);
    // Actually run the summation
    *sum_h = 0.0f;
    assert(hipMemset(sum_d, 0.0f, 1) == hipSuccess);
    CubDebugExit(hipcub::DeviceReduce::Sum(cub_tmp_storage, cub_tmp_bytes, lat, sum_d, M_count));
    // Retreive the result
    assert(hipMemcpy(sum_h, sum_d, sizeof(float), hipMemcpyDeviceToHost) == hipSuccess);
    // Reset the lattice to zero for the next run
    assert(hipMemset(lat, 0.0f, M_bytes) == hipSuccess);
    // Print the result
    fprintf(stderr, "%llu: Mean = %f\n", k, *sum_h / (N0*Ni*Ni*Ni));
  }

  // Free resources
  delete_lattice(&lat);
  delete_rng(&states);
  hipFree(sum_d);
  free(sum_h);
  CubDebugExit(g_allocator.DeviceFree(cub_tmp_storage));
}

__host__ int main() {

  mc_mean(8, 5000, 1.0f, true, 1000);

  return 0;
}
